#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include "Node.h"
#include "SOM.h"
#include <time.h>
#include "constants.h"
#include "resource.h"
#include <stdio.h>
#include <omp.h>
#include <sstream>
#include <fstream>
#include <iostream>
#include "hiprand.h"

//the width of the matrix (not the number of total elements)
int N = 1600;
//grid and block size
dim3 block(16,16);
dim3 grid(N/block.x, N/block.y);

float cpuStart;
float cpuEnd;
char* g_szApplicationName = "Kohonen Self Organizing Map Demo";
char* g_szWindowClassName = "MyWindowClass";
vector<vector<double>> m_TrainingSet;

void CreateDataSet();
//pointer to a Self Organising Map
SOM*  pSOM = new SOM();
//the data for the training

//used to create the back buffer
static HDC   hdcBackBuffer;
static HBITMAP hBitmap;
static HBITMAP hOldBitmap;

LRESULT CALLBACK WindowProc(HWND   hwnd,
	UINT   msg,
	WPARAM wParam,
	LPARAM lParam)
{

	//these hold the dimensions of the client window area
	static int CxClient, CyClient;

	//used to create the back buffer
	//static HDC   hdcBackBuffer;
	//static HBITMAP hBitmap;
	//static HBITMAP hOldBitmap;


	switch (msg)
	{

	case WM_CREATE:
	{
		//to get get the size of the client window first we need  to create
		//a RECT and then ask Windows to fill in our RECT structure with
		//the client window size. Then we assign to CxClient and CyClient 
		//accordingly
		RECT rect;

		GetClientRect(hwnd, &rect);

		CxClient = rect.right;
		CyClient = rect.bottom;

		//seed random number generator
		srand((unsigned)time(NULL));

		//create a memory device context
		hdcBackBuffer = CreateCompatibleDC(NULL);

		//get the DC for the front buffer
		HDC hdc = GetDC(hwnd);

		hBitmap = CreateCompatibleBitmap(hdc,
			CxClient,
			CyClient);


		//select the bitmap into the memory device context
		hOldBitmap = (HBITMAP)SelectObject(hdcBackBuffer, hBitmap);

		//don't forget to release the DC
		ReleaseDC(hwnd, hdc);

		pSOM->Create(CxClient, CyClient, constNumCellsAcross,
			constNumCellsDown, constNumIterations);
		
	}

		break;
	case WM_KEYUP:
	{
		switch (wParam)
		{
		case VK_ESCAPE:
		{
			SendMessage(hwnd, WM_DESTROY, NULL, NULL);

			PostQuitMessage(0);
		}

			break;

		case 'R':
		{
			delete pSOM;

			pSOM = new SOM();
			pSOM->Create(CxClient, CyClient, constNumCellsAcross,
				constNumCellsDown, constNumIterations);
		}

			break;
		}
	}


	case WM_PAINT:
	{

		PAINTSTRUCT ps;

		BeginPaint(hwnd, &ps);

		//fill the backbuffer with white
		BitBlt(hdcBackBuffer,
			0,
			0,
			CxClient,
			CyClient,
			NULL,
			NULL,
			NULL,
			WHITENESS);

		pSOM->Render(hdcBackBuffer);


		//now blit the backbuffer to the front
		BitBlt(ps.hdc, 0, 0, CxClient, CyClient, hdcBackBuffer, 0, 0, SRCCOPY);

		EndPaint(hwnd, &ps);

	}

		break;

		//has the user resized the client area?
	case WM_SIZE:
	{
		//if so we need to update our variables so that any drawing
		//we do using cxClient and cyClient is scaled accordingly
		CxClient = LOWORD(lParam);
		CyClient = HIWORD(lParam);

		//now to resize the backbuffer accordingly. First select
		//the old bitmap back into the DC
		SelectObject(hdcBackBuffer, hOldBitmap);

		//don't forget to do this or you will get resource leaks
		DeleteObject(hBitmap);

		//get the DC for the application
		HDC hdc = GetDC(hwnd);

		//create another bitmap of the same size and mode
		//as the application
		hBitmap = CreateCompatibleBitmap(hdc,
			CxClient,
			CyClient);

		ReleaseDC(hwnd, hdc);

		//select the new bitmap into the DC
		SelectObject(hdcBackBuffer, hBitmap);

	}

		break;

	case WM_DESTROY:
	{

		//clean up our backbuffer objects
		SelectObject(hdcBackBuffer, hOldBitmap);

		DeleteDC(hdcBackBuffer);
		DeleteObject(hBitmap);

		// kill the application, this sends a WM_QUIT message  
		PostQuitMessage(0);
	}

		break;

	}//end switch

	//this is where all the messages not specifically handled by our 
	//winproc are sent to be processed
	return DefWindowProc(hwnd, msg, wParam, lParam);
}
wchar_t *convertCharArrayToLPCWSTR(const char* charArray)
{
	wchar_t* wString = new wchar_t[4096];
	MultiByteToWideChar(CP_ACP, 0, charArray, -1, wString, 4096);
	return wString;
}

string Convert(float number){
    std::ostringstream buff;
    buff<<number;
    return buff.str();   
}


__device__ int	  RandInt(int x, int y) { return 8 % (y - x + 1) + x; }

__device__ Node* device_findBestMatchingNode(const vector<double> &vec, vector<Node> mSOM)
{
	Node* winner = NULL;

	double LowestDistance = 999999;

	for (int n = 0; n<mSOM.size(); ++n)
	{
		double dist = mSOM[n].CalculateDistance(vec);

		if (dist < LowestDistance)
		{
			LowestDistance = dist;

			winner = &mSOM[n];
		}
	}

	return winner;
}
__global__ void device_epoch(vector<vector<double>> data, int numIterations, vector<Node> mSOM,
	int mapRadius, int iterationCount, int timeConstant, int learningRate){
	
	//enter the training loop
	if (--numIterations > 0)
	{
		//the input vectors are presented to the network at random
		int ThisVector = RandInt(0, data.size() - 1);

		//present the vector to each node and determine the BMU
		Node* winningNode = device_findBestMatchingNode(data[ThisVector], mSOM);

		//calculate the width of the neighbourhood for this timestep
		int neighbourhoodRadius = mapRadius * exp(-(double)iterationCount / timeConstant);

		//Now to adjust the weight vector of the BMU and its
		//neighbours

		//For each node calculate the m_dInfluence (Theta from equation 6 in
		//the tutorial. If it is greater than zero adjust the node's weights
		//accordingly
		for (int n = 0; n<mSOM.size(); ++n)
		{
			//calculate the Euclidean distance (squared) to this node from the
			//BMU
			double DistToNodeSq = (winningNode->X() - mSOM[n].X()) *
				(winningNode->X() - mSOM[n].X()) +
				(winningNode->Y() - mSOM[n].Y()) *
				(winningNode->Y() - mSOM[n].Y());

			double WidthSq = neighbourhoodRadius * neighbourhoodRadius;

			//if within the neighbourhood adjust its weights
			if (DistToNodeSq < (neighbourhoodRadius * neighbourhoodRadius))
			{

				//calculate by how much its weights are adjusted
				int influence = exp(-(DistToNodeSq) / (2 * WidthSq));

				mSOM[n].AdjustWeights(data[ThisVector],
					learningRate,
					influence);
			}

		}//next node


		//reduce the learning rate
		learningRate = constStartLearningRate * exp(-(double)iterationCount / numIterations);

		++iterationCount;

	}

	else
	{
		//bDone = true;
	
	}
	//return true
}


int WINAPI WinMain(HINSTANCE hInstance,
	HINSTANCE hPrevInstance,
	LPSTR     szCmdLine,
	int       iCmdShow)
{
	CreateDataSet();
	//handle to our window
	HWND           hWnd;

	//our window class structure
	WNDCLASSEX     winclass;

	// first fill in the window class stucture
	winclass.cbSize = sizeof(WNDCLASSEX);
	winclass.style = CS_HREDRAW | CS_VREDRAW;
	winclass.lpfnWndProc = WindowProc;
	winclass.cbClsExtra = 0;
	winclass.cbWndExtra = 0;
	winclass.hInstance = hInstance;
	winclass.hIcon = LoadIcon(hInstance, MAKEINTRESOURCE(IDI_ICON1));
	winclass.hCursor = LoadCursor(NULL, IDC_ARROW);
	winclass.hbrBackground = NULL;
	winclass.lpszMenuName = NULL;
	winclass.lpszClassName = g_szWindowClassName;
	winclass.hIconSm = LoadIcon(hInstance, MAKEINTRESOURCE(IDI_ICON1));

	//register the window class
	if (!RegisterClassEx(&winclass))
	{
		MessageBox(NULL, TEXT("Registration Failed!"), TEXT("Error"), 0);

		//exit the application
		return 0;
	}

	//create a window with the *client* area specified.
	RECT rect;
	rect.left = 0;
	rect.top = 0;
	rect.bottom = constWindowHeight;
	rect.right = constWindowWidth;

	if (!AdjustWindowRectEx(&rect, CS_HREDRAW | CS_VREDRAW, true, NULL))
	{
		MessageBox(NULL, TEXT("Problem creating window"), TEXT("error!"), MB_OK);
		return 0;
	}

	//create the window and assign its ID to hwnd    
	hWnd = CreateWindowEx(NULL,                 // extended style
		g_szWindowClassName,  // window class name
		g_szApplicationName,  // window caption
		WS_OVERLAPPED | WS_VISIBLE | WS_CAPTION | WS_SYSMENU,
		GetSystemMetrics(SM_CXSCREEN) / 2 - constWindowWidth / 2,
		GetSystemMetrics(SM_CYSCREEN) / 2 - constWindowHeight / 2,
		rect.right,           // initial x size
		rect.bottom,          // initial y size
		NULL,                 // parent window handle
		NULL,                 // window menu handle
		hInstance,            // program instance handle
		NULL);                // creation parameters

	//make sure the window creation has gone OK
	if (!hWnd)
	{
		MessageBox(NULL, TEXT("CreateWindowEx Failed!"), TEXT("Error!"), 0);
	}

	//make the window visible
	ShowWindow(hWnd, iCmdShow);
	UpdateWindow(hWnd);

	// enter the message loop
	bool bDone = false;
	MSG msg;


			int numIterations =  constNumIterations;
			int mapRadius = max(constWindowWidth, constWindowHeight) / 2;
			int timeConstant = numIterations / log(mapRadius);
			int learningRate = constStartLearningRate;
			int iterationCount = 0;
			vector<Node> mSOM = pSOM->getSOM();
	cpuStart= omp_get_wtime();

	while (!bDone)
	{
		while (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE))
		{
			if (msg.message == WM_QUIT)
			{
				// Stop loop if it's a quit message
				bDone = true;
			}

			else
			{
				TranslateMessage(&msg);
				DispatchMessage(&msg);
			}
		}

		if (!pSOM->FinishedTraining())//if still training map
		{
			

			device_epoch<<<grid,block>>>(m_TrainingSet, numIterations, mSOM, mapRadius, iterationCount, timeConstant,learningRate);
			
			/*vector<vector<double>> data, int numIterations,  vector<Node> mSOM, int mapRadius, 
					int iterationCount, int timeConstant, int learningRate*/


			//this will call WM_PAINT which will render the map
			
			InvalidateRect(hWnd, NULL, TRUE);
			UpdateWindow(hWnd);
		}

		if(pSOM->getIteration() == constNumIterations){
			cpuEnd =  omp_get_wtime();	
			bDone = true; 
		}
	}

		float cpuTime = (cpuEnd-cpuStart);//*1000;

		char str[256];
		char num[256];
		char it[256];
		sprintf_s(str, " CPU Time: %6f \n", cpuTime);
		sprintf_s(num, " Number of Nodes: %d \n",pSOM->getSize());
		sprintf_s(it, " Number of Iterations: %d \n",pSOM->getIteration());

		OutputDebugString("----------CPU TIME-----------\n");
		OutputDebugString(str);
		OutputDebugString("----------Number of Nodes-----------\n");
		OutputDebugString(num);
		OutputDebugString("----------Number of Iterations-----------\n");
		OutputDebugString(it);
		OutputDebugString("-----------------------------\n");
	


	delete pSOM;

	UnregisterClass (g_szWindowClassName, winclass.hInstance);

	return msg.wParam;
}


void Render(HDC surface)
{
	pSOM->Render(surface);
}

inline double RandFloat()		   { return (rand()) / (RAND_MAX + 1.0); }

void CreateDataSet()
{

#ifndef RANDOM_TRAINING_SETS

	//create a data set
	vector<double> red, green, blue, yellow, orange, purple, dk_green, dk_blue;
	//push to back of vector 
	red.push_back(1);
	red.push_back(0);
	red.push_back(0);

	green.push_back(0);
	green.push_back(1);
	green.push_back(0);

	dk_green.push_back(0);
	dk_green.push_back(0.5);
	dk_green.push_back(0.25);

	blue.push_back(0);
	blue.push_back(0);
	blue.push_back(1);

	dk_blue.push_back(0);
	dk_blue.push_back(0);
	dk_blue.push_back(0.5);

	yellow.push_back(1);
	yellow.push_back(1);
	yellow.push_back(0.2);

	orange.push_back(1);
	orange.push_back(0.4);
	orange.push_back(0.25);

	purple.push_back(1);
	purple.push_back(0);
	purple.push_back(1);

	m_TrainingSet.push_back(red);
	m_TrainingSet.push_back(green);
	m_TrainingSet.push_back(blue);
	m_TrainingSet.push_back(yellow);
	m_TrainingSet.push_back(orange);
	m_TrainingSet.push_back(purple);
	m_TrainingSet.push_back(dk_green);
	m_TrainingSet.push_back(dk_blue);


#else

	//choose a random number of training sets
	int NumSets = RandInt(constMinNumTrainingSets, constMaxNumTrainingSets);

	for (int s = 0; s<NumSets; ++s)
	{

		vector<double> set;

		set.push_back(RandFloat());
		set.push_back(RandFloat());
		set.push_back(RandFloat());

		m_TrainingSet.push_back(set);
	}

#endif
}